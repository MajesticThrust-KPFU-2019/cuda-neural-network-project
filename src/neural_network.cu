#include "hip/hip_runtime.h"
#include "neural_network.cuh"

#include <initializer_list>
#include <vector>
#include <exception>
#include <random>
#include <algorithm>
#include <functional>
#include <time.h>

#include <hip/hip_runtime.h>
#include <hipblas.h>

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include "cuda_helpers/hip/hip_runtime_api.h"

#include "utils.hpp"
#include <iostream>

// TODO remove
void print_vector2(const std::vector<float>& v) {
	for (auto o : v) {
		std::cout << o << " ";
	}
	std::cout << std::endl;
}

/**
 * Invoke a __global__ function with max occupancy.
 * @param dynamicSMemSize - Per-block dynamic shared memory usage intended, in bytes (see hipOccupancyMaxPotentialBlockSize)
 * @param blockSizeLimit - The maximum block size func is designed to work with. 0 means no limit ((see hipOccupancyMaxPotentialBlockSize)
 * @param inputSize - number of elements that can be processed in parallel by a single thread each
 * @param func - the __global__ function in question
 * @param funcArgs - func arguments as a parameter_pack
 */
template<class ...Args>
void cudaInvokeMaxOccupancy(size_t dynamicSMemSize, int blockSizeLimit,
		int inputSize, void (*func)(Args...), Args ... funcArgs) {
	// see https://developer.nvidia.com/blog/cuda-pro-tip-occupancy-api-simplifies-launch-configuration/
	// The launch configurator returned block size
	int blockSize;
	// The minimum grid size needed to achieve the
	// maximum occupancy for a full device launch
	int minGridSize;

	hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, func,
			dynamicSMemSize, blockSizeLimit);

	// The actual grid size needed, based on input size
	// Round up according to array size
	auto gridSize = (inputSize + blockSize - 1) / blockSize;

	func<<< gridSize, blockSize >>>(funcArgs...);
}

/**
 * Convert 2-dim index to 1-dim. Used to index matrices stored as 1-dim arrays.
 * @param i
 * @param j
 * @param ld - leading dimension; number of rows for column-major storage
 */
inline unsigned int IDX2C(unsigned int i, unsigned int j, unsigned int ld) {
	return (((j) * (ld)) + (i));
}

const decltype(NeuralNetwork::layer_sizes_)& NeuralNetwork::layer_sizes() const {
	return this->layer_sizes_;
}

void print_cuda_matrix(const float* dev_ptr, size_t nrows, size_t ncols) {
	auto h_matrix = new float[nrows * ncols];

	hipMemcpy(h_matrix, dev_ptr, nrows * ncols * sizeof(float),
			hipMemcpyDeviceToHost);
	getLastCudaError(
			string_format("Unable to copy data from device (%d) to host",
					dev_ptr).c_str());

	for (auto i = 0; i < nrows; i++) {
		for (auto j = 0; j < ncols; j++) {
			auto idx = IDX2C(i, j, nrows);
			printf("%.6f\t", h_matrix[idx]);
		}
		printf("\n");
	}

	delete[] h_matrix;
}

NeuralNetwork::NeuralNetwork(std::initializer_list<unsigned int> layer_sizes) :
		layer_sizes_(layer_sizes) {
	if (this->layer_sizes_.size() < 2) {
		throw std::invalid_argument("Must specify at least 2 layers!");
	}

	// allocate weights
	this->dev_weights = std::vector<float*>(this->layer_sizes_.size() - 1);
	for (auto i = 0; i < this->layer_sizes_.size() - 1; i++) {
		auto prev_ls = this->layer_sizes_[i];
		auto next_ls = this->layer_sizes_[i + 1];

		auto size = next_ls * prev_ls * sizeof(float);
		hipMalloc((void**) &(this->dev_weights[i]), size);
		getLastCudaError(
				string_format("Error allocating weights %d", i).c_str());
	}

	// allocate biases
	this->dev_biases = std::vector<float*>(this->layer_sizes_.size() - 1);
	for (auto i = 0; i < this->layer_sizes_.size() - 1; i++) {
		auto next_ls = this->layer_sizes_[i + 1];

		auto size = next_ls * sizeof(float);
		hipMalloc((void**) &(this->dev_biases[i]), size);
		getLastCudaError(
				string_format("Error allocating biases %d", i).c_str());
	}

	// allocate activation and error vectors
	// skip the input layer
	this->dev_activations = std::vector<float*>(this->layer_sizes_.size() - 1);
	this->dev_errors = std::vector<float*>(this->layer_sizes_.size() - 1);
	for (auto i = 0; i < this->layer_sizes_.size() - 1; i++) {
		auto layer_size = this->layer_sizes_[i + 1];

		auto size = layer_size * sizeof(float);
		hipMalloc((void**) &(this->dev_activations[i]), size);
		getLastCudaError(
				string_format("Error allocating activation %d", i).c_str());

		hipMalloc((void**) &(this->dev_errors[i]), size);
		getLastCudaError(string_format("Error allocating error %d", i).c_str());
	}

	// allocate intermediate vector
	auto max_layer_size = *std::max_element(this->layer_sizes_.cbegin(),
			this->layer_sizes_.cend());
	hipMalloc((void**) &this->dev_intermediate,
			max_layer_size * sizeof(float));
	getLastCudaError(
			string_format("Error allocating intermediate vector").c_str());

	// allocate context
	auto status = hipblasCreate(&this->cublasHandle);
	checkCudaErrors(status);
}

NeuralNetwork::~NeuralNetwork() {
	for (auto const &dev_ptr : this->dev_weights) {
		hipFree(dev_ptr);
	}

	for (auto const &dev_ptr : this->dev_biases) {
		hipFree(dev_ptr);
	}

	for (auto const &dev_ptr : this->dev_activations) {
		hipFree(dev_ptr);
	}

	for (auto const &dev_ptr : this->dev_errors) {
		hipFree(dev_ptr);
	}

	hipFree(this->dev_intermediate);

	auto status = hipblasDestroy(this->cublasHandle);
	checkCudaErrors(status);
}

void NeuralNetwork::init_random(float min, float max) {
//	std::random_device r;
//	std::mt19937 eng(r()); // a source of random data
	std::mt19937 eng(time(nullptr));

	std::uniform_real_distribution<float> dist(min, max);
	auto gen_f = bind(dist, eng);

	// fill weights in a random fashion
	for (auto i = 0; i < this->layer_sizes_.size() - 1; i++) {
		auto l_prev = this->layer_sizes_[i];
		auto l_next = this->layer_sizes_[i + 1];

		auto weights_size = l_next * l_prev * sizeof(float);
		auto biases_size = l_next * sizeof(float);

		printf(
				"\nFilling layer %d; sizes: [%d, %d]; weights size: %d, biases size: %d\n",
				i, l_prev, l_next, weights_size, biases_size);

		// fill weights with (min .. max] floats
		auto weights = new std::vector<float>(weights_size);
		std::generate(weights->begin(), weights->end(), gen_f);
		printf("Generated weights %d on host:\n", i);
		print_vector2(*weights);

		auto biases = new std::vector<float>(biases_size);
		std::generate(biases->begin(), biases->end(), gen_f);
		printf("Generated biases %d on host:\n", i);
		print_vector2(*biases);

		// copy to device
		hipMemcpy(this->dev_weights[i], weights->data(), weights_size,
				hipMemcpyHostToDevice);
		getLastCudaError(
				string_format("Copy random weights %d to device", i).c_str());

		hipMemcpy(this->dev_biases[i], biases->data(), biases_size,
				hipMemcpyHostToDevice);
		getLastCudaError(
				string_format("Copy random biases %d to device", i).c_str());

		delete weights;
		delete biases;
	}
}

//	void init_from_data() {
//		// TODO init from data how? what format?
//	}

/**
 * Vectorized sigmoid function.
 * @param x - input vector allocated on device
 * @param y - output vector allocated on device
 * @param N - length of each vector
 */
__global__ void sigmoid(const float *x, float *y, unsigned int N) {
	int tidx = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;

	for (; tidx < N; tidx += stride) {
		y[tidx] = 1.0 / (1 + expf(-x[tidx]));
	}
}

/**
 * Vectorized sigmoid derivative function.
 * @param x - input vector allocated on device with precomputed sigmoid values
 * @param y - output vector allocated on device
 * @param N - length of each vector
 */
__global__ void sigmoid_derivative(const float *sig_x, float *y,
		unsigned int N) {
	int tidx = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;

	for (; tidx < N; tidx += stride) {
		y[tidx] = sig_x[tidx] * (1 - sig_x[tidx]);
	}
}

/**
 * Haramard product for vectors. y is both an input and an output.
 */
__global__ void vhadamard(const float *x, float *y, unsigned int N) {
	int tidx = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;

	for (; tidx < N; tidx += stride) {
		y[tidx] = y[tidx] * x[tidx];
	}
}

void NeuralNetwork::evaluate(const float *dev_input) {
	// TODO rewrite to use batches? (switch Sgemv to Sgemm)
	hipblasStatus_t status;

	const float *layer_input = dev_input;

	float alpha = 1, beta = 1;

	// propagate forward
	for (auto i = 0; i < this->layer_sizes_.size() - 1; i++) {
		printf("\nIteration %d\n\n", i);

		auto l_prev = this->layer_sizes_[i];
		auto l_next = this->layer_sizes_[i + 1];

		printf("Prev layer - %d, next layer - %d\n\n", l_prev, l_next);

		printf("Weights %d:\n", i);
		print_cuda_matrix(this->dev_weights[i], l_next, l_prev);
		printf("\n");

		printf("Biases %d:\n", i);
		print_cuda_matrix(this->dev_biases[i], l_next, 1);
		printf("\n");

		printf("Input %d:\n", i);
		print_cuda_matrix(layer_input, l_prev, 1);
		printf("\n");

		auto layer_output = this->dev_activations[i];

		// y param in hipblasSgemv is both an input (bias) and an output (activation)
		// hence, copy bias into the activation
		hipMemcpy(layer_output, this->dev_biases[i], l_next * sizeof(float),
				hipMemcpyDeviceToDevice);
		getLastCudaError(
				string_format("Copying bias %d to activation", i).c_str());

		status = hipblasSgemv(this->cublasHandle, HIPBLAS_OP_N, l_next, l_prev,
				&alpha, this->dev_weights[i], l_next, layer_input, 1, &beta,
				layer_output, 1);
		checkCudaErrors(status);

		printf("Output %d after Sgemv:\n", i);
		print_cuda_matrix(layer_output, l_next, 1);
		printf("\n");

		// apply sigmoid in-place to activation vector
		cudaInvokeMaxOccupancy(0, 0, l_next, sigmoid,
				(const float *) layer_output, layer_output, l_next);

		printf("Output %d:\n", i);
		print_cuda_matrix(layer_output, l_next, 1);
		printf("\n");

		layer_input = layer_output;
	}

	// copy final activation to the output
//	auto last_layer_size = this->layer_sizes_.back();
//	auto last_activation = this->dev_activations.back();
//	hipMemcpy(dev_output, last_activation, last_layer_size * sizeof(float),
//			hipMemcpyDeviceToDevice);
//	getLastCudaError("Unable to copy the last ANN activation to the output");
}

void NeuralNetwork::train(const float* dev_x_train, const float* dev_y_train,
		float learning_rate, float *out_cost) {
	hipblasStatus_t status;
	// needed to pass into hipblasSgemm as a negative coefficient when updating weights
	learning_rate = -learning_rate;

	// see https://brilliant.org/wiki/backpropagation/
	// ^ The Backpropagation Algorithm paragraph

	int i = this->layer_sizes_.size() - 1 - 1;
	auto size = this->layer_sizes_[i];

	// save layer outputs
	this->evaluate(dev_x_train);

	// TODO biases

	// compute error for the output layer

	// write sigmoid derivative into error vector
	cudaInvokeMaxOccupancy(0, 0, size, sigmoid_derivative,
			(const float *) this->dev_activations[i], this->dev_errors[i],
			size);

	// compute output delta and overwrite output layer activation
	// (which is no longer needed, as the sigmoid derivative is already computed)
	float alpha = -1;
	status = hipblasSaxpy(this->cublasHandle, size, &alpha, dev_y_train, 1,
			this->dev_activations[i], 1);
	checkCudaErrors(status);

	// write MSE error to the method output
	status = hipblasSdot(this->cublasHandle, size, this->dev_activations[i], 1,
			this->dev_activations[i], 1, out_cost);
	checkCudaErrors(status);
	*out_cost /= 2.0 * size;

	printf("MSE error = %f\n", *out_cost);
	print_cuda_matrix(this->dev_activations[i], size, 1);

	// compute output layer error
	cudaInvokeMaxOccupancy(0, 0, size, vhadamard,
			(const float *) this->dev_activations[i], this->dev_errors[i],
			size);

	printf("Last layer error:\n");
	print_cuda_matrix(this->dev_errors[i], size, 1);

	i--;
	printf("Starting loop from i = %d; i >= 0 == %d\n", i, i >= 0);

	// compute errors for hidden layers, update weights
//	while (i >= 0) {
	while (true) {
		printf("Backprop i = %d", i);
		auto rnext = this->layer_sizes_[i + 1];
		auto rprev = this->layer_sizes_[i];

		// update weights
		float beta = 1;
		status = hipblasSgemm(this->cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N,
				rnext, rprev, 1, &learning_rate, this->dev_errors[i + 1], rnext,
				this->dev_activations[i], rprev, &beta,
				this->dev_weights[i + 1], rnext);
		checkCudaErrors(status);

		// compute sigmoid derivative and write it into the output vector
		cudaInvokeMaxOccupancy(0, 0, rprev, sigmoid_derivative,
				(const float *) this->dev_activations[i],
				this->dev_activations[i], rprev);

		// write error intermediate into the error vector
		alpha = 1;
		status = hipblasSgemv(this->cublasHandle, HIPBLAS_OP_T, rnext, rprev,
				&alpha, this->dev_weights[i + 1], rnext,
				this->dev_errors[i + 1], 1, 0, this->dev_errors[i], 1);
		checkCudaErrors(status);

		// compute error
		cudaInvokeMaxOccupancy(0, 0, rprev, vhadamard,
				(const float *) this->dev_activations[i], this->dev_errors[i],
				rprev);

		i--;

		if (i >= 0)
			break;
	}
}

