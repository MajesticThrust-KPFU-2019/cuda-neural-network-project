#include "hip/hip_runtime.h"
#include "neural_network.cuh"

#include <initializer_list>
#include <vector>
#include <exception>
#include <random>
#include <algorithm>
#include <functional>
#include <time.h>

#include <hip/hip_runtime.h>
#include <hipblas.h>

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include "cuda_helpers/hip/hip_runtime_api.h"

#include "utils.hpp"
#include <iostream>

// TODO remove
template<typename T>
void print_vector2(const std::vector<T>& v) {
	for (auto o : v) {
		std::cout << o << " ";
	}
	std::cout << std::endl;
}

/**
 * Invoke a __global__ function with max occupancy.
 * @param dynamicSMemSize - Per-block dynamic shared memory usage intended, in bytes (see hipOccupancyMaxPotentialBlockSize)
 * @param blockSizeLimit - The maximum block size func is designed to work with. 0 means no limit ((see hipOccupancyMaxPotentialBlockSize)
 * @param inputSize - number of elements that can be processed in parallel by a single thread each
 * @param func - the __global__ function in question
 * @param funcArgs - func arguments as a parameter_pack
 */
template<class ...Args>
void cudaInvokeMaxOccupancy(size_t dynamicSMemSize, int blockSizeLimit,
		int inputSize, void (*func)(Args...), Args ... funcArgs) {
	// see https://developer.nvidia.com/blog/cuda-pro-tip-occupancy-api-simplifies-launch-configuration/
	// The launch configurator returned block size
	int blockSize;
	// The minimum grid size needed to achieve the
	// maximum occupancy for a full device launch
	int minGridSize;

	hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, func,
			dynamicSMemSize, blockSizeLimit);

	// The actual grid size needed, based on input size
	// Round up according to array size
	auto gridSize = (inputSize + blockSize - 1) / blockSize;

	printf("Launching kernel with gridSize=%d, blockSize=%d\n", gridSize,
			blockSize);

	func<<< gridSize, blockSize >>>(funcArgs...);
}

/**
 * Convert 2-dim index to 1-dim. Used to index matrices stored as 1-dim arrays.
 * @param i
 * @param j
 * @param ld - leading dimension; number of rows for column-major storage
 */
inline unsigned int IDX2C(unsigned int i, unsigned int j, unsigned int ld) {
	return (((j) * (ld)) + (i));
}

const decltype(NeuralNetwork::layer_sizes_)& NeuralNetwork::layer_sizes() const {
	return this->layer_sizes_;
}

void print_cuda_matrix(const float* dev_ptr, size_t nrows, size_t ncols) {
	auto h_matrix = new float[nrows * ncols];

	hipMemcpy(h_matrix, dev_ptr, nrows * ncols * sizeof(float),
			hipMemcpyDeviceToHost);
	getLastCudaError(
			string_format("Unable to copy data from device (%d) to host",
					dev_ptr).c_str());

	for (auto i = 0; i < nrows; i++) {
		for (auto j = 0; j < ncols; j++) {
			auto idx = IDX2C(i, j, nrows);
			printf("%.6f\t", h_matrix[idx]);
		}
		printf("\n");
	}

	delete[] h_matrix;
}

NeuralNetwork::NeuralNetwork(std::initializer_list<unsigned int> layer_sizes) :
		layer_sizes_(layer_sizes) {
	if (this->layer_sizes_.size() < 2) {
		throw std::invalid_argument("Must specify at least 2 layers!");
	}

	// allocate weights
	this->dev_weights = std::vector<float*>(this->layer_sizes_.size() - 1);
	for (auto i = 0; i < this->layer_sizes_.size() - 1; i++) {
		auto prev_ls = this->layer_sizes_[i];
		auto next_ls = this->layer_sizes_[i + 1];

		auto size = next_ls * prev_ls * sizeof(float);
		hipMalloc((void**) &(this->dev_weights[i]), size);
		getLastCudaError(
				string_format("Error allocating weights %d", i).c_str());
	}

	// allocate biases
	this->dev_biases = std::vector<float*>(this->layer_sizes_.size() - 1);
	for (auto i = 0; i < this->layer_sizes_.size() - 1; i++) {
		auto next_ls = this->layer_sizes_[i + 1];

		auto size = next_ls * sizeof(float);
		hipMalloc((void**) &(this->dev_biases[i]), size);
		getLastCudaError(
				string_format("Error allocating biases %d", i).c_str());
	}

	// allocate activations
	this->dev_activations = std::vector<float*>(this->layer_sizes_.size());
	for (auto i = 0; i < this->layer_sizes_.size(); i++) {
		auto layer_size = this->layer_sizes_[i];

		auto size = layer_size * sizeof(float);
		hipMalloc((void**) &(this->dev_activations[i]), size);
		getLastCudaError(
				string_format("Error allocating activation %d", i).c_str());
	}

	// allocate error vectors, skip the input layer
	this->dev_errors = std::vector<float*>(this->layer_sizes_.size() - 1);
	for (auto i = 0; i < this->layer_sizes_.size() - 1; i++) {
		auto layer_size = this->layer_sizes_[i + 1];

		auto size = layer_size * sizeof(float);
		hipMalloc((void**) &(this->dev_errors[i]), size);
		getLastCudaError(string_format("Error allocating error %d", i).c_str());
	}

	// allocate intermediate vector
	auto max_layer_size = *std::max_element(this->layer_sizes_.cbegin(),
			this->layer_sizes_.cend());
	hipMalloc((void**) &this->dev_intermediate,
			max_layer_size * sizeof(float));
	getLastCudaError(
			string_format("Error allocating intermediate vector").c_str());

	// allocate context
	auto status = hipblasCreate(&this->cublasHandle);
	checkCudaErrors(status);
}

NeuralNetwork::~NeuralNetwork() {
	for (auto const &dev_ptr : this->dev_weights) {
		hipFree(dev_ptr);
	}

	for (auto const &dev_ptr : this->dev_biases) {
		hipFree(dev_ptr);
	}

	for (auto const &dev_ptr : this->dev_activations) {
		hipFree(dev_ptr);
	}

	for (auto const &dev_ptr : this->dev_errors) {
		hipFree(dev_ptr);
	}

	hipFree(this->dev_intermediate);

	auto status = hipblasDestroy(this->cublasHandle);
	checkCudaErrors(status);
}

void NeuralNetwork::init_random(float min, float max) {
//	std::random_device r;
//	std::mt19937 eng(r()); // a source of random data
	std::mt19937 eng(time(nullptr));

	std::uniform_real_distribution<float> dist(min, max);
	auto gen_f = bind(dist, eng);

	// fill weights in a random fashion
	for (auto i = 0; i < this->layer_sizes_.size() - 1; i++) {
		auto l_prev = this->layer_sizes_[i];
		auto l_next = this->layer_sizes_[i + 1];

		auto weights_size = l_next * l_prev * sizeof(float);
		auto biases_size = l_next * sizeof(float);

		printf(
				"\nFilling layer %d; sizes: [%d, %d]; weights size: %d, biases size: %d\n",
				i, l_prev, l_next, weights_size, biases_size);

		// fill weights with (min .. max] floats
		auto weights = new std::vector<float>(weights_size);
		std::generate(weights->begin(), weights->end(), gen_f);
		printf("Generated weights %d on host:\n", i);
		print_vector2(*weights);

		auto biases = new std::vector<float>(biases_size);
		std::generate(biases->begin(), biases->end(), gen_f);
		printf("Generated biases %d on host:\n", i);
		print_vector2(*biases);

		// copy to device
		hipMemcpy(this->dev_weights[i], weights->data(), weights_size,
				hipMemcpyHostToDevice);
		getLastCudaError(
				string_format("Copy random weights %d to device", i).c_str());

		hipMemcpy(this->dev_biases[i], biases->data(), biases_size,
				hipMemcpyHostToDevice);
		getLastCudaError(
				string_format("Copy random biases %d to device", i).c_str());

		delete weights;
		delete biases;
	}
}

//	void init_from_data() {
//		// TODO init from data how? what format?
//	}

/**
 * Vectorized sigmoid function.
 * @param x - input vector allocated on device
 * @param y - output vector allocated on device
 * @param N - length of each vector
 */
__global__ void sigmoid(const float *x, float *y, unsigned int N) {
	int tidx = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;

	for (; tidx < N; tidx += stride) {
		y[tidx] = 1.0 / (1 + expf(-x[tidx]));
	}
}

/**
 * Vectorized sigmoid derivative function.
 * @param x - input vector allocated on device with precomputed sigmoid values
 * @param y - output vector allocated on device
 * @param N - length of each vector
 */
__global__ void sigmoid_derivative(const float *sig_x, float *y,
		unsigned int N) {
	int tidx = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;

	for (; tidx < N; tidx += stride) {
		y[tidx] = sig_x[tidx] * (1 - sig_x[tidx]);
	}
}

/**
 * Hadamard product for vectors. y is both an input and an output.
 */
__global__ void vhadamard(const float *x, float *y, unsigned int N) {
	int tidx = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;

	for (; tidx < N; tidx += stride) {
		y[tidx] = y[tidx] * x[tidx];
	}
}

void NeuralNetwork::evaluate(const float *dev_input) {
	// TODO rewrite to use batches? (switch Sgemv to Sgemm)
	hipblasStatus_t status;

//	const float *layer_input = dev_input;
	hipMemcpy(this->dev_activations[0], dev_input,
			this->layer_sizes_[0] * sizeof(float), hipMemcpyDeviceToDevice);

	float alpha = 1, beta = 1;

	// propagate forward
	for (auto i = 0; i < this->layer_sizes_.size() - 1; i++) {
		printf("\nIteration %d\n\n", i);

		auto l_prev = this->layer_sizes_[i];
		auto l_next = this->layer_sizes_[i + 1];

		auto layer_input = this->dev_activations[i];
		auto layer_output = this->dev_activations[i + 1];

		printf("Prev layer - %d, next layer - %d\n\n", l_prev, l_next);

		printf("Weights %d:\n", i);
		print_cuda_matrix(this->dev_weights[i], l_next, l_prev);
		printf("\n");

		printf("Biases %d:\n", i);
		print_cuda_matrix(this->dev_biases[i], l_next, 1);
		printf("\n");

		printf("Input %d:\n", i);
		print_cuda_matrix(layer_input, l_prev, 1);
		printf("\n");

		// y param in hipblasSgemv is both an input (bias) and an output (activation)
		// hence, copy bias into the activation
		hipMemcpy(layer_output, this->dev_biases[i], l_next * sizeof(float),
				hipMemcpyDeviceToDevice);
		getLastCudaError(
				string_format("Copying bias %d to activation", i).c_str());

		status = hipblasSgemv(this->cublasHandle, HIPBLAS_OP_N, l_next, l_prev,
				&alpha, this->dev_weights[i], l_next, layer_input, 1, &beta,
				layer_output, 1);
		checkCudaErrors(status);

		printf("Output %d after Sgemv:\n", i);
		print_cuda_matrix(layer_output, l_next, 1);
		printf("\n");

		// apply sigmoid in-place to activation vector
		cudaInvokeMaxOccupancy(0, 0, l_next, sigmoid,
				(const float *) layer_output, layer_output, l_next);

		printf("Output %d:\n", i);
		print_cuda_matrix(layer_output, l_next, 1);
		printf("\n");
	}

	// copy final activation to the output
//	auto last_layer_size = this->layer_sizes_.back();
//	auto last_activation = this->dev_activations.back();
//	hipMemcpy(dev_output, last_activation, last_layer_size * sizeof(float),
//			hipMemcpyDeviceToDevice);
//	getLastCudaError("Unable to copy the last ANN activation to the output");
}

void NeuralNetwork::train(const float* dev_x_train, const float* dev_y_train,
		float learning_rate, float *out_cost) {
	hipblasStatus_t status;
	// needed to pass into hipblasSgemm as a negative coefficient when updating weights
	learning_rate = -learning_rate;

	// see https://brilliant.org/wiki/backpropagation/
	// ^ The Backpropagation Algorithm paragraph

	int i = this->layer_sizes_.size() - 1 - 1;	// last index - 1

	// save layer outputs
	this->evaluate(dev_x_train);

	auto rnext = this->layer_sizes_[i + 1];
	auto rprev = this->layer_sizes_[i];
	printf("NN train; rnext = %d, rprev = %d\n", rnext, rprev);
	auto next_activation = this->dev_activations[i + 1];
	auto prev_activation = this->dev_activations[i];

	// TODO biases

	// compute error for the output layer

	// write sigmoid derivative into error vector
	cudaInvokeMaxOccupancy(0, 0, rnext, sigmoid_derivative,
			(const float *) next_activation, this->dev_errors[i], rnext);

	printf("Last layer sigmoid derivative:\n");
	print_cuda_matrix(this->dev_errors[i], rnext, 1);

	// compute output delta and overwrite output layer activation
	// (which is no longer needed, as the sigmoid derivative is already computed)
	float alpha = -1;
	status = hipblasSaxpy(this->cublasHandle, rnext, &alpha, dev_y_train, 1,
			next_activation, 1);
	checkCudaErrors(status);

	// write MSE error to the method output
	status = hipblasSdot(this->cublasHandle, rnext, next_activation, 1,
			next_activation, 1, out_cost);
	checkCudaErrors(status);
	*out_cost /= 2.0 * rnext;

	printf("MSE error = %f\n", *out_cost);
	print_cuda_matrix(next_activation, rnext, 1);

	// compute output layer error
	cudaInvokeMaxOccupancy(0, 0, rnext, vhadamard,
			(const float *) next_activation, this->dev_errors[i], rnext);

	printf("Last layer error:\n");
	print_cuda_matrix(this->dev_errors[i], rnext, 1);

	printf("Updating weights\n");
	printf("Layers:\n");
	print_vector2(this->layer_sizes());
	printf("Error i+1:\n");
	print_cuda_matrix(this->dev_errors[i], rnext, 1);
	printf("Activation i:\n");
	print_cuda_matrix(prev_activation, 1, rprev);
	printf("Current weights i\n");
	print_cuda_matrix(this->dev_weights[i], rnext, rprev);

	// print weights delta
//	float* testMem;
//	hipMalloc((void**) &testMem, sizeof(float) * rnext * rprev);
//	float beta = 0;
//	status = hipblasSgemm(this->cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, rnext,
//			rprev, 1, &learning_rate, this->dev_errors[i], rnext,
//			this->dev_activations[i - 1], 1, &beta, testMem, rnext);
//	checkCudaErrors(status);
//	printf("Weights delta (w/o learning rate)\n");
//	print_cuda_matrix(testMem, rnext, rprev);
//	hipFree(testMem);

	// update weights
	float beta = 1;
	status = hipblasSgemm(this->cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, rnext,
			rprev, 1, &learning_rate, this->dev_errors[i], rnext,
			prev_activation, 1, &beta, this->dev_weights[i], rnext);
	checkCudaErrors(status);

	printf("Updated weights\n");
	print_cuda_matrix(this->dev_weights[i], rnext, rprev);

	// print error vectors; TODO remove
//	for (auto j = 0; j < this->layer_sizes_.rnext() - 1; j++) {
//		printf("Layer %d with %d nodes\nError vector\n", j + 1,
//				this->layer_sizes_[j + 1]);
//		print_cuda_matrix(this->dev_errors[j], 1, this->layer_sizes_[j + 1]);
//	}

	// compute errors for hidden layers, update weights
	for (--i; i >= 0; i--) {
		auto rnext2 = this->layer_sizes_[i + 2];
		rnext = this->layer_sizes_[i + 1];
		rprev = this->layer_sizes_[i];
		next_activation = this->dev_activations[i + 1];
		prev_activation = this->dev_activations[i];
		printf("Backprop i = %d; rnext2 = %d, rnext = %d, rprev = %d\n", i,
				rnext2, rnext, rprev);

		// compute sigmoid derivative and write it into the output vector
		cudaInvokeMaxOccupancy(0, 0, rnext, sigmoid_derivative,
				(const float *) next_activation, next_activation, rnext);

		printf("Sigmoid derivative\n");
		print_cuda_matrix(next_activation, rnext, 1);

		printf("Error i + 1\n");
		print_cuda_matrix(this->dev_errors[i + 1], 1, rnext2);

		// write error intermediate into the error vector
		alpha = 1;
		float intermediate_beta = 0;
		status = hipblasSgemv(this->cublasHandle, HIPBLAS_OP_T, rnext2, rnext,
				&alpha, this->dev_weights[i + 1], rnext2,
				this->dev_errors[i + 1], 1, &intermediate_beta,
				this->dev_errors[i], 1);
		checkCudaErrors(status);

		printf("Error intermediate\n");
		print_cuda_matrix(this->dev_errors[i], rnext, 1);

		// compute error
		cudaInvokeMaxOccupancy(0, 0, rnext, vhadamard,
				(const float *) next_activation, this->dev_errors[i], rnext);

		printf("Error\n");
		print_cuda_matrix(this->dev_errors[i], rnext, 1);

		printf("Current weights i\n");
		print_cuda_matrix(this->dev_weights[i], rnext, rprev);

		// update weights
		status = hipblasSgemm(this->cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N,
				rnext, rprev, 1, &learning_rate, this->dev_errors[i], rnext,
				prev_activation, 1, &beta, this->dev_weights[i], rnext);
		checkCudaErrors(status);

		printf("Updated weights\n");
		print_cuda_matrix(this->dev_weights[i], rnext, rprev);
	}
}

