#include "hip/hip_runtime.h"
#include "neural_network.cuh"

#include <initializer_list>
#include <vector>
#include <exception>

#include <hip/hip_runtime.h>
#include "cuda_helpers/hip/hip_runtime_api.h"
#include <hipblas.h>

#include "utils.hpp"

//void printLastCublasError(hipblasStatus_t status, const char *msg) {
//	if (status != HIPBLAS_STATUS_SUCCESS)
//		printf(stderr, "%s\n%s\n", msg, _cudaGetErrorEnum(status));
//}

/**
 * Convert 2-dim index to 1-dim. Used to index matrices stored as 1-dim arrays.
 * @param i
 * @param j
 * @param ld - leading dimension; number of rows for column-major storage
 */
inline unsigned int IDX2C(unsigned int i, unsigned int j, unsigned int ld) {
	return (((j) * (ld)) + (i));
}

const decltype(NeuralNetwork::layer_sizes_)& NeuralNetwork::layer_sizes() const {
	return this->layer_sizes_;
}

NeuralNetwork::NeuralNetwork(std::initializer_list<unsigned int> layer_sizes) :
		layer_sizes_(layer_sizes) {
	if (this->layer_sizes_.size() < 2) {
		throw std::invalid_argument("Must specify at least 2 layers!");
	}

	// allocate weights + biases
	this->dev_weights = std::vector<float*>(this->layer_sizes_.size() - 1);
	for (auto i = 1; i < this->layer_sizes_.size(); i++) {
		auto prev_ls = this->layer_sizes_[i - 1];
		auto next_ls = this->layer_sizes_[i];
		// include biases in weights
		auto size = next_ls * (prev_ls + 1) + sizeof(float);
		auto w_i = i - 1;
		hipMalloc((void**) &(this->dev_weights[w_i]), size);
		getLastCudaError(
				string_format("Error allocating weights %d", w_i).c_str());
	}

	// allocate activation vectors
	this->dev_activations = std::vector<float*>(this->layer_sizes_.size() - 1);
	for (auto i = 1; i < this->layer_sizes_.size(); i++) {
		auto layer_size = this->layer_sizes_[i];

		// include trailing 1 for a bias
		auto size = (layer_size + 1) * sizeof(float);
		auto act_i = i - 1;
		hipMalloc((void**) &(this->dev_activations[act_i]), size);
		getLastCudaError(
				string_format("Error allocating activation %d", act_i).c_str());

	}

	// allocate context
	auto status = hipblasCreate(&this->cublasHandle);
	checkCudaErrors(status);
}

NeuralNetwork::~NeuralNetwork() {
	for (auto const &devPtr : this->dev_weights) {
		hipFree(devPtr);
	}

	for (auto const &devPtr : this->dev_activations) {
		hipFree(devPtr);
	}

	auto status = hipblasDestroy(this->cublasHandle);
	checkCudaErrors(status);
}

__global__ void fill_array_randomly() {
	// TODO
}

void NeuralNetwork::init_random() {
	// fill weights in a random fashion
	for (auto i = 0; i < this->layer_sizes_.size(); i++) {
		auto l_prev = this->layer_sizes_[i];
		auto l_next = this->layer_sizes_[i + 1];
	}

}

//	void init_from_data() {
//		// TODO init from data how? what format?
//	}

/**
 * Forward propagation. Fills internal activation vectors.
 *
 * Expects the input to be a vector with the same length as the input layer.
 * Expects the output to have enough allocated space for the output vector.
 */
void NeuralNetwork::predict(float *devInput, float *devOutput) {
	hipblasStatus_t status;

	float *layerInput = devInput;
	float *layerOutput = this->dev_activations[0];

	float alpha = 1, beta = 1;

	// propagate forward
	for (auto i = 0; i < this->layer_sizes_.size(); i++) {
		auto l_prev = this->layer_sizes_[i];
		auto l_next = this->layer_sizes_[i + 1];

		status = hipblasSgemv(this->cublasHandle, HIPBLAS_OP_N, l_next,
				l_prev + 1, &alpha, this->dev_weights[i], l_next, layerInput, 1,
				&beta, layerOutput, 1);
		checkCudaErrors(status);

		layerInput = layerOutput;
		layerOutput = this->dev_activations[i + 1];
	}

	// copy final activation to the output
	auto last_i = this->layer_sizes_.size() - 1;
	hipMemcpy(devOutput, this->dev_activations[last_i], this->layer_sizes_[last_i], hipMemcpyDeviceToDevice);
	getLastCudaError("Unable to copy the last ANN activation to the output");
}

// pass dev pointer(s) to the batch, and a dev pointer for output
void NeuralNetwork::train_batch() {
	//
}

